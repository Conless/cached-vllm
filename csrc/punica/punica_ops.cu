#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <c10/cuda/CUDAGuard.h>
#include <cstdint>

#include "type_convert.h"
#include "../cuda_compat.h"
#include "bgmv/bgmv_config.h"
#include "sgmv/sgmv.h"
#include "sgmv_flashinfer/sgmv_config.h"


//====== utils ======

inline void check_shape(const torch::Tensor &a, const torch::Tensor &b,
                        const char *a_name, const char *b_name) {
  TORCH_CHECK(a.dim() == b.dim(), a_name, ".dim() != ", b_name, ".dim(). ",
              a.dim(), " vs ", b.dim());
  for (int i = 0; i < a.dim(); ++i) {
    TORCH_CHECK(a.size(i) == b.size(i), a_name, ".size(", i, ") != ", b_name,
                ".size(", i, ")");
  }
}

inline constexpr uint64_t pack_u32(uint32_t a, uint32_t b) {
  return (uint64_t(a) << 32) | uint64_t(b);
}

#define CHECK_CUDA(x) TORCH_CHECK(x.is_cuda(), #x " must be a CUDA tensor")

#define CHECK_CONTIGUOUS(x)                                                    \
  TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")

#define CHECK_INPUT(x)                                                         \
  CHECK_CUDA(x);                                                               \
  CHECK_CONTIGUOUS(x)

#define CHECK_DIM(d, x)                                                        \
  TORCH_CHECK(x.dim() == d, #x " must be a " #d "D tensor")

#define CHECK_SHAPE(a, b) check_shape(a, b, #a, #b)

#define CHECK_EQ(a, b)                                                         \
  TORCH_CHECK(a == b, "CHECK_EQ(" #a ", " #b ") failed. ", a, " vs ", b)

#define CHECK_GE(a, b) \
  TORCH_CHECK((a) >= (b), "CHECK_GE(" #a ", " #b ") failed. ", a, " vs ", b)

//====== dispatch pytorch dtype ======

#define _DISPATCH_SWITCH(cond, ...) \
  [&]() -> bool {                   \
    switch (cond) {                 \
      __VA_ARGS__                   \
      default:                      \
        return false;               \
    }                               \
  }()

#define _DISPATCH_DTYPE_CASE(enum_type, c_type_, ...) \
  case enum_type: {                                   \
    using c_type = c_type_;                           \
    return __VA_ARGS__();                             \
  }

#define _DISPATCH_DTYPE_CASES(...)                                 \
  _DISPATCH_DTYPE_CASE(at::ScalarType::Half, nv_half, __VA_ARGS__) \
  _DISPATCH_DTYPE_CASE(at::ScalarType::BFloat16, hip_bfloat16, __VA_ARGS__)

#define DISPATCH_TORCH_DTYPE(scalar_type, ...) \
  _DISPATCH_SWITCH(scalar_type, _DISPATCH_DTYPE_CASES(__VA_ARGS__))

//====== bgmv ======

template <typename in_T, typename out_T, typename W_T>
inline bool launch_bgmv_kernel(out_T *Y, const in_T *X, const W_T *W,
                               const int64_t *lora_indices,
                               uint32_t in_features, uint32_t out_features,
                               int64_t y_offset, int64_t full_y_size,
                               int64_t batch_size, int64_t num_layers,
                               int64_t layer_idx, float scale) {
  // NOTE(woosuk): While Punica supports various combinations of input/output
  // data types, we limit the supported data types to reduce the binary size.
  constexpr bool is_input_float = std::is_same<in_T, float>::value;
  constexpr bool is_output_float = std::is_same<out_T, float>::value;
  if (is_input_float) {
    if (!std::is_same<out_T, W_T>::value) {
      return false;
    }
  } else if (is_output_float) {
    if (!std::is_same<in_T, W_T>::value) {
      return false;
    }
  } else if (!(std::is_same<in_T, W_T>::value &&
               std::is_same<out_T, W_T>::value)) {
    return false;
  }

  switch (pack_u32(in_features, out_features)) {
#define CASE_ONESIDE(_in_T, _out_T, _W_T, feat_in, feat_out)                   \
  case pack_u32(feat_in, feat_out):                                            \
    bgmv_kernel<feat_in, feat_out>(Y, X, W, lora_indices, y_offset,            \
                                   full_y_size, batch_size, num_layers,        \
                                   layer_idx, scale);                          \
    break;
#define CASE(_in_T, _out_T, _W_T, narrow, wide)                                \
  CASE_ONESIDE(in_T, out_T, W_T, narrow, wide)                                 \
  CASE_ONESIDE(in_T, out_T, W_T, wide, narrow)

    FOR_BGMV_WIDE_NARROW(CASE, _, _, _)
    FOR_INST_BGMV_WIDE_NARROW(CASE_ONESIDE, _, _, _)
#undef CASE
#undef CASE_ONESIDE
  default:
    return false;
  }
  return true;
}

void dispatch_bgmv(torch::Tensor y, torch::Tensor x, torch::Tensor w,
                   torch::Tensor indicies, int64_t layer_idx, float scale) {
  CHECK_INPUT(y);
  CHECK_INPUT(x);
  CHECK_INPUT(w);
  CHECK_INPUT(indicies);

  CHECK_DIM(2, y);
  CHECK_DIM(2, x);
  CHECK_DIM(4, w);
  CHECK_DIM(1, indicies);

  int64_t B = x.size(0);
  int64_t h_in = x.size(1);
  int64_t h_out = y.size(1);
  int64_t num_layers = w.size(1);
  CHECK_EQ(w.size(3), h_in);
  CHECK_EQ(w.size(2), h_out);
  CHECK_EQ(indicies.size(0), x.size(0));
  CHECK_EQ(y.size(0), x.size(0));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(x));
  bool ok = false;
  if (h_in <= 128512 && h_out <= 128512) {
    // TODO: See if we can get rid of this massive nested switch
    switch (x.scalar_type()) {
    case at::ScalarType::Half:
      switch (y.scalar_type()) {
      case at::ScalarType::Half:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<nv_half *>(y.data_ptr()),
                                  static_cast<nv_half *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<nv_half *>(y.data_ptr()),
                                  static_cast<nv_half *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      case at::ScalarType::BFloat16:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<hip_bfloat16 *>(y.data_ptr()),
                                  static_cast<nv_half *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<hip_bfloat16 *>(y.data_ptr()),
                                  static_cast<nv_half *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      case at::ScalarType::Float:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<float *>(y.data_ptr()),
                                  static_cast<nv_half *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<float *>(y.data_ptr()),
                                  static_cast<nv_half *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      default:
        break;
      }
      break;
    case at::ScalarType::BFloat16:
      switch (y.scalar_type()) {
      case at::ScalarType::Half:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<nv_half *>(y.data_ptr()),
                                  static_cast<hip_bfloat16 *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<nv_half *>(y.data_ptr()),
                                  static_cast<hip_bfloat16 *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      case at::ScalarType::BFloat16:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<hip_bfloat16 *>(y.data_ptr()),
                                  static_cast<hip_bfloat16 *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<hip_bfloat16 *>(y.data_ptr()),
                                  static_cast<hip_bfloat16 *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      case at::ScalarType::Float:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<float *>(y.data_ptr()),
                                  static_cast<hip_bfloat16 *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<float *>(y.data_ptr()),
                                  static_cast<hip_bfloat16 *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      default:
        break;
      }
      break;
    case at::ScalarType::Float:
      switch (y.scalar_type()) {
      case at::ScalarType::Half:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<nv_half *>(y.data_ptr()),
                                  static_cast<float *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<nv_half *>(y.data_ptr()),
                                  static_cast<float *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      case at::ScalarType::BFloat16:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<hip_bfloat16 *>(y.data_ptr()),
                                  static_cast<float *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<hip_bfloat16 *>(y.data_ptr()),
                                  static_cast<float *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      case at::ScalarType::Float:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<float *>(y.data_ptr()),
                                  static_cast<float *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<float *>(y.data_ptr()),
                                  static_cast<float *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out, 0,
                                  h_out, B, num_layers, layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      default:
        break;
      }
      break;
    default:
      break;
    }
  }
  TORCH_CHECK(ok, "No suitable kernel.", " h_in=", h_in, " h_out=", h_out,
              " dtype=", x.scalar_type(), " out_dtype=", y.scalar_type());
}

void dispatch_bgmv_low_level(torch::Tensor y, torch::Tensor x, torch::Tensor w,
                             torch::Tensor indicies, int64_t layer_idx,
                             float scale, int64_t h_in, int64_t h_out,
                             int64_t y_offset) {
  CHECK_INPUT(y);
  CHECK_INPUT(x);
  CHECK_INPUT(w);
  CHECK_INPUT(indicies);

  CHECK_DIM(2, y);
  CHECK_DIM(2, x);
  CHECK_DIM(4, w);
  CHECK_DIM(1, indicies);

  int64_t B = x.size(0);
  int64_t num_layers = w.size(1);
  int64_t full_y_size = y.size(1);
  CHECK_EQ(w.size(3), h_in);
  CHECK_EQ(w.size(2), h_out);
  CHECK_EQ(indicies.size(0), x.size(0));
  CHECK_EQ(y.size(0), x.size(0));
  const at::cuda::OptionalCUDAGuard device_guard(device_of(x));
  bool ok = false;
  if (h_in <= 128512 && h_out <= 128512) {
    // TODO: See if we can get rid of this massive nested switch
    switch (x.scalar_type()) {
    case at::ScalarType::Half:
      switch (y.scalar_type()) {
      case at::ScalarType::Half:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<nv_half *>(y.data_ptr()),
                                  static_cast<nv_half *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<nv_half *>(y.data_ptr()),
                                  static_cast<nv_half *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      case at::ScalarType::BFloat16:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<hip_bfloat16 *>(y.data_ptr()),
                                  static_cast<nv_half *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<hip_bfloat16 *>(y.data_ptr()),
                                  static_cast<nv_half *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      case at::ScalarType::Float:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<float *>(y.data_ptr()),
                                  static_cast<nv_half *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<float *>(y.data_ptr()),
                                  static_cast<nv_half *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      default:
        break;
      }
      break;
    case at::ScalarType::BFloat16:
      switch (y.scalar_type()) {
      case at::ScalarType::Half:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<nv_half *>(y.data_ptr()),
                                  static_cast<hip_bfloat16 *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<nv_half *>(y.data_ptr()),
                                  static_cast<hip_bfloat16 *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      case at::ScalarType::BFloat16:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<hip_bfloat16 *>(y.data_ptr()),
                                  static_cast<hip_bfloat16 *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<hip_bfloat16 *>(y.data_ptr()),
                                  static_cast<hip_bfloat16 *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      case at::ScalarType::Float:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<float *>(y.data_ptr()),
                                  static_cast<hip_bfloat16 *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<float *>(y.data_ptr()),
                                  static_cast<hip_bfloat16 *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      default:
        break;
      }
      break;
    case at::ScalarType::Float:
      switch (y.scalar_type()) {
      case at::ScalarType::Half:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<nv_half *>(y.data_ptr()),
                                  static_cast<float *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<nv_half *>(y.data_ptr()),
                                  static_cast<float *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      case at::ScalarType::BFloat16:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<hip_bfloat16 *>(y.data_ptr()),
                                  static_cast<float *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<hip_bfloat16 *>(y.data_ptr()),
                                  static_cast<float *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      case at::ScalarType::Float:
        switch (w.scalar_type()) {
        case at::ScalarType::Half:
          ok = launch_bgmv_kernel(static_cast<float *>(y.data_ptr()),
                                  static_cast<float *>(x.data_ptr()),
                                  static_cast<nv_half *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        case at::ScalarType::BFloat16:
          ok = launch_bgmv_kernel(static_cast<float *>(y.data_ptr()),
                                  static_cast<float *>(x.data_ptr()),
                                  static_cast<hip_bfloat16 *>(w.data_ptr()),
                                  indicies.data_ptr<int64_t>(), h_in, h_out,
                                  y_offset, full_y_size, B, num_layers,
                                  layer_idx, scale);
          break;
        default:
          break;
        }
        break;
      default:
        break;
      }
      break;
    default:
      break;
    }
  }
  TORCH_CHECK(ok, "No suitable kernel.", " h_in=", h_in, " h_out=", h_out,
              " dtype=", x.scalar_type(), " out_dtype=", y.scalar_type());
}

//====== sgmv ======

void dispatch_sgmv_cutlass(torch::Tensor y, torch::Tensor x,
                           torch::Tensor w_ptr, torch::Tensor s,
                           torch::Tensor tmp, int layer_idx) {
  CHECK_INPUT(y);
  CHECK_INPUT(x);
  CHECK_INPUT(w_ptr);
  CHECK_INPUT(s);
  CHECK_INPUT(tmp);

  CHECK_DIM(2, y);
  CHECK_DIM(2, x);
  CHECK_DIM(1, w_ptr);
  CHECK_DIM(1, s);
  CHECK_DIM(1, tmp);

  int num_problems = s.size(0) - 1;
  int d_in = x.size(1);
  int d_out = y.size(1);
  CHECK_EQ(tmp.size(0), static_cast<int64_t>(sgmv_tmp_size(num_problems)));
  hipStream_t stream = c10::cuda::getCurrentCUDAStream();
  bool ok = DISPATCH_TORCH_DTYPE(x.scalar_type(), [&] {
    return sgmv<c_type>((c_type*)y.data_ptr(), (c_type*)x.data_ptr(),
                        (c_type**)w_ptr.data_ptr(), s.data_ptr<int32_t>(),
                        tmp.data_ptr<uint8_t>(), num_problems, d_in, d_out,
                        layer_idx, stream);
  });
  TORCH_CHECK(ok, "No suitable kernel.", " dtype=", x.scalar_type());
}

void dispatch_sgmv_shrink(torch::Tensor y, torch::Tensor x, torch::Tensor w_ptr,
                          torch::Tensor s, torch::Tensor tmp, int layer_idx) {
  CHECK_INPUT(y);
  CHECK_INPUT(x);
  CHECK_INPUT(w_ptr);
  CHECK_INPUT(s);
  CHECK_INPUT(tmp);

  CHECK_DIM(2, y);
  CHECK_DIM(2, x);
  CHECK_DIM(1, w_ptr);
  CHECK_DIM(1, s);
  CHECK_DIM(1, tmp);

  uint32_t num_problems = s.size(0) - 1;
  uint32_t d_in = x.size(1);
  uint32_t d_out = y.size(1);
  CHECK_EQ(tmp.scalar_type(), at::ScalarType::Byte);
  CHECK_EQ(tmp.size(0), 8 * 1024 * 1024);
  hipStream_t stream = c10::cuda::getCurrentCUDAStream();

#define CASE(_T, D_OUT)                                    \
  case D_OUT:                                              \
    return sgmv_shrink<c_type, D_OUT>(                     \
        (c_type*)y.data_ptr(), (c_type*)x.data_ptr(),      \
        (c_type**)w_ptr.data_ptr(), s.data_ptr<int32_t>(), \
        tmp.data_ptr<uint8_t>(), num_problems, d_in, layer_idx, stream);

  bool ok = DISPATCH_TORCH_DTYPE(x.scalar_type(), [&] {
    switch (d_out) {
      FOR_SGMV_NARROW(CASE, c_type);
      default:
        return false;
    }
  });

#undef CASE
  TORCH_CHECK(ok, "No suitable kernel.", " dtype=", x.scalar_type(),
              " d_out=", d_out);
}